#include "hip/hip_runtime.h"
/* Copyright 2018 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "taso/ops.h"
#include "taso/cuda_helper.h"
using namespace taso;

Model::Model()
: isTraining(false), print_cost(false)
{
  //int* a = (int*) malloc(sizeof(int) * 8);
  checkCUDA(hipSetDevice(0));
  checkCUDNN(hipdnnCreate(&dnn));
  checkCUDA(hipblasCreate(&blas));
  workSpaceSize = WORK_SPACE_SIZE;
  global_unique_id = 100;
  checkCUDA(hipMalloc(&workSpace, workSpaceSize));
  // printf("handle.workSpace = 0x%x\n", workSpace);
  // create all descriptors
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&biasTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&scaleTensor));
  checkCUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
  checkCUDNN(hipdnnCreatePoolingDescriptor(&poolDesc));
  checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
  checkCUDNN(hipdnnCreateOpTensorDescriptor(&opDesc));
  // allocate tensors for measuring performance
  checkCUDA(hipMalloc(&inputPtr, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&biasPtr, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&outputPtr, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&filterPtr, MAX_TENSOR_SIZE));
  // create tensors for batch norm
  checkCUDA(hipMalloc(&scalePtr, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&runningMean, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&runningVar, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&saveMean, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&saveVar, MAX_TENSOR_SIZE));
  // create cuda events
  checkCUDA(hipEventCreate(&startEvent));
  checkCUDA(hipEventCreate(&endEvent));
}

float Model::measure_oplist_runtime(const std::vector<OpBase*>& opBaseList)
{
  const int num_runs = 100;
  // warmup
  for (int times = 0; times < num_runs; times++)
    for (int i = 0; i < opBaseList.size(); i++)
      opBaseList[i]->forward();
  // measure runtime
  // checkCUDA(hipDeviceSynchronize());
  checkCUDA(hipEventRecord(startEvent));
  for (int times = 0; times < num_runs; times++) {
    for (int i = 0; i < opBaseList.size(); i++)
      opBaseList[i]->forward();
  }
  checkCUDA(hipEventRecord(endEvent));
  checkCUDA(hipEventSynchronize(endEvent));
  float milliseconds;
  hipEventElapsedTime(&milliseconds, startEvent, endEvent);
  return milliseconds / num_runs;
}

void* Model::allocate_memory(size_t size, const DATATYPE* data_initial)
{
  void* ptr;
  if (size == 0) {
    // Note: Special value for zero-sized tensor
    ptr = (void*) 0x1;
  } else {
    checkCUDA(hipMalloc(&ptr, size));
  }
  if (data_initial != NULL) {
    checkCUDA(hipMemcpy(ptr, data_initial, size, hipMemcpyDefault));
  }
  return ptr;
}

bool Model::copy_memory(DATATYPE* dst, const DATATYPE* src, size_t size)
{
  checkCUDA(hipMemcpy(dst, src, size, hipMemcpyDefault));
  return true;
}
