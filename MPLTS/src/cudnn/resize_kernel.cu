/* Copyright 2019 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "taso/ops.h"
#include "taso/cuda_helper.h"
using namespace taso;

void Resize::map(void)
{
  // allocate tensors
  size_t outputSize = sizeof(DATATYPE) * outputs[0].volume();
  checkCUDA(hipMalloc(&outputs[0].data_ptr, outputSize));
}

void Resize::unmap(void)
{
  checkCUDA(hipFree(outputs[0].data_ptr));
}

void Resize::forward(bool block)
{
  if (block)
    checkCUDA(hipDeviceSynchronize());
}

void Model::measure_resize_cost(Resize* resize)
{
  // FIXME: assume the cost is zero for now
  resize->runtime = 0;
}
