/* Copyright 2019 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "taso/ops.h"
#include "taso/cuda_helper.h"
using namespace taso;

void Reshape::map(void)
{
  // allocate tensors
  size_t outputSize = sizeof(DATATYPE) * outputs[0].volume();
  checkCUDA(hipMalloc(&outputs[0].data_ptr, outputSize));
}

void Reshape::unmap(void)
{
  checkCUDA(hipFree(outputs[0].data_ptr));
}

void Reshape::forward(bool block)
{
  if (block)
    checkCUDA(hipDeviceSynchronize());
}

void Model::measure_reshape_cost(Reshape* reshape)
{
  // FIXME: assume the cost is zero for now
  reshape->runtime = 0;
}
