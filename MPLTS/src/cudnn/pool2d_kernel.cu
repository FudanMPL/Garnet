#include "hip/hip_runtime.h"
/* Copyright 2018 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "taso/ops.h"
#include "taso/cuda_helper.h"
using namespace taso;

void Pool2D::map(void)
{
  // create descriptors
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreatePoolingDescriptor(&poolDesc));
  int inputN = inputs[0].dim[0];
  int inputC = inputs[0].dim[1];
  int inputH = inputs[0].dim[2];
  int inputW = inputs[0].dim[3];
  int padH, padW;
  get_padding(&padH, &padW);
  // set descriptors
  checkCUDNN(hipdnnSetTensor4dDescriptor(inputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, inputN, inputC, inputH, inputW));
  hipdnnPoolingMode_t mode;
  if (type == OP_POOL2D_MAX)
    mode = HIPDNN_POOLING_MAX;
  else if (type == OP_POOL2D_AVG)
    mode = HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING;
  checkCUDNN(hipdnnSetPooling2dDescriptor(poolDesc, mode, HIPDNN_PROPAGATE_NAN,
      kernelH, kernelW, padH, padW, strideH, strideW));
  int n, c, h, w;
  checkCUDNN(hipdnnGetPooling2dForwardOutputDim(poolDesc, 
      inputTensor, &n, &c, &h, &w));
  assert(n == inputN);
  assert(c == inputC);
  assert(outputs[0].dim[2] == h);
  assert(outputs[0].dim[3] == w);
  checkCUDNN(hipdnnSetTensor4dDescriptor(outputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, n, c, h, w));
  if (activation != AC_MODE_NONE) {
    checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
    hipdnnActivationMode_t mode = get_activation_mode(activation);
    checkCUDNN(hipdnnSetActivationDescriptor(actiDesc, mode,
        HIPDNN_PROPAGATE_NAN, 0.0));
  }
  // allocate tensors
  size_t outputSize = sizeof(DATATYPE) * n * c * h * w;
  checkCUDA(hipMalloc(&outputs[0].data_ptr, outputSize));
}

void Pool2D::unmap(void)
{
  checkCUDNN(hipdnnDestroyTensorDescriptor(inputTensor));
  checkCUDNN(hipdnnDestroyTensorDescriptor(outputTensor));
  checkCUDNN(hipdnnDestroyPoolingDescriptor(poolDesc));
  if (activation != AC_MODE_NONE) {
    checkCUDNN(hipdnnDestroyActivationDescriptor(actiDesc));
  }
  // free tensors
  checkCUDA(hipFree(outputs[0].data_ptr));
}

void Pool2D::forward(bool block)
{
  const float alpha = 1.0f;
  const float beta = 0.0f;
  checkCUDNN(hipdnnPoolingForward(model->dnn, poolDesc,
      &alpha, inputTensor, inputs[0].data_ptr,
      &beta, outputTensor, outputs[0].data_ptr));
  if (activation != AC_MODE_NONE) {
    checkCUDNN(hipdnnActivationForward(model->dnn, actiDesc,
        &alpha, outputTensor, outputs[0].data_ptr,
        &beta, outputTensor, outputs[0].data_ptr));
  }
  if (block)
    checkCUDA(hipDeviceSynchronize());
}

void Model::measure_pool2d_cost(Pool2D* pool)
{
  const float alpha = 1.0f;
  const float beta = 0.0f;
  int inputN = pool->inputs[0].dim[0];
  int inputC = pool->inputs[0].dim[1];
  int inputH = pool->inputs[0].dim[2];
  int inputW = pool->inputs[0].dim[3];
  int outputH = pool->outputs[0].dim[2];
  int outputW = pool->outputs[0].dim[3];
  int padH, padW;
  // Reference: https://www.tensorflow.org/api_guides/python/nn#Convolution
  switch (pool->padding) {
    case PD_MODE_SAME:
      int totalPadH, totalPadW;
      if (inputH % pool->strideH == 0)
        totalPadH = max(pool->kernelH - pool->strideH, 0);
      else
        totalPadH = max(pool->kernelH - (inputH % pool->strideH), 0);
      if (inputW % pool->strideW == 0)
        totalPadW = max(pool->kernelW - pool->strideW, 0);
      else
        totalPadW = max(pool->kernelW - (inputW % pool->strideW), 0);
      // assert same padding on both sides
      padH = (totalPadH + 1) / 2;
      padW = (totalPadW + 1)/ 2;
      break;
    case PD_MODE_VALID:
      padH = 0;
      padW = 0;
      break;
    default:
      assert(false);
  }
  checkCUDNN(hipdnnSetTensor4dDescriptor(inputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, inputN, inputC, inputH, inputW));
  hipdnnPoolingMode_t mode;
  if (pool->type == OP_POOL2D_MAX)
    mode = HIPDNN_POOLING_MAX;
  else if (pool->type == OP_POOL2D_AVG)
    mode = HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING;
  checkCUDNN(hipdnnSetPooling2dDescriptor(poolDesc, mode,
      HIPDNN_PROPAGATE_NAN, pool->kernelH, pool->kernelW, padH, padW,
      pool->strideH, pool->strideW));
  checkCUDNN(hipdnnSetActivationDescriptor(actiDesc, HIPDNN_ACTIVATION_RELU,
        HIPDNN_NOT_PROPAGATE_NAN, 0.0));
  int n, c, h, w;
  checkCUDNN(hipdnnGetPooling2dForwardOutputDim(poolDesc,
      inputTensor, &n, &c, &h, &w));
  assert(n == inputN);
  assert(c == inputC);
  assert(outputH == h);
  assert(outputW == w);
  checkCUDNN(hipdnnSetTensor4dDescriptor(outputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, n, c, h, w));
  size_t inputSize = sizeof(DATATYPE) * inputN * inputC * inputH * inputW;
  size_t outputSize = sizeof(DATATYPE) * inputN * inputC * outputH * outputW;
  assert(inputSize < MAX_TENSOR_SIZE);
  assert(outputSize < MAX_TENSOR_SIZE);
  checkCUDA(hipDeviceSynchronize());
  for (int i = 0; i < WARMUP_TIMES + REPEAT_TIMES; i++) {
    if (i == WARMUP_TIMES) {
      checkCUDA(hipEventRecord(startEvent));
    }
    checkCUDNN(hipdnnPoolingForward(dnn, poolDesc,
        &alpha, inputTensor, inputPtr,
        &beta, outputTensor, outputPtr));
    if (pool->activation != AC_MODE_NONE) {
      checkCUDNN(hipdnnActivationForward(dnn, actiDesc,
          &alpha, outputTensor, outputPtr,
          &beta, outputTensor, outputPtr));
    }
  }
  checkCUDA(hipEventRecord(endEvent));
  checkCUDA(hipEventSynchronize(endEvent));
  float milliseconds;
  hipEventElapsedTime(&milliseconds, startEvent, endEvent);
  pool->runtime = milliseconds / REPEAT_TIMES;
  if (print_cost)
    printf("  measure[Pool2D]: i(%d %d %d %d) k(%d %d) s(%d %d) p(%d %d) cost(%.4lf)\n",
           inputN, inputC, inputH, inputW, pool->kernelH, pool->kernelW,
           pool->strideH, pool->strideW, padH, padW, pool->runtime);
}

