#include "hip/hip_runtime.h"
/* Copyright 2018 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "taso/ops.h"
#include "taso/cuda_helper.h"
using namespace taso;

void Conv2D::map(void)
{
  // create descriptors
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&biasTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
  int inputN = inputs[0].dim[0];
  int inputC = inputs[0].dim[1];
  int inputH = inputs[0].dim[2];
  int inputW = inputs[0].dim[3];
  int outputC = inputs[1].dim[0];
  int groups = inputs[0].dim[1] / inputs[1].dim[1];
  int padH, padW;
  get_padding(&padH, &padW);
  // set descriptors
  checkCUDNN(hipdnnSetTensor4dDescriptor(inputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, inputN, inputC, inputH, inputW));
  checkCUDNN(hipdnnSetTensor4dDescriptor(biasTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, 1, outputC, 1, 1));
  checkCUDNN(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT,
      HIPDNN_TENSOR_NCHW, inputs[1].dim[0], inputs[1].dim[1],
      inputs[1].dim[2], inputs[1].dim[3]));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc, padH, padW,
      strideH, strideW, 1/*dilationH*/, 1/*dilationW*/,
      HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
  checkCUDNN(hipdnnSetConvolutionMathType(convDesc, HIPDNN_TENSOR_OP_MATH));
  if (groups != 1) {
    checkCUDNN(hipdnnSetConvolutionGroupCount(convDesc, groups));
  }
  int n, c, h, w;
  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc,
      inputTensor, filterDesc, &n, &c, &h, &w));
  assert(n == inputN);
  assert(c == outputC);
  assert(outputs[0].dim[2] == h);
  assert(outputs[0].dim[3] == w);
  checkCUDNN(hipdnnSetTensor4dDescriptor(outputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, n, c, h, w));
  if (activation != AC_MODE_NONE) {
    checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
    hipdnnActivationMode_t mode = get_activation_mode(activation);
    checkCUDNN(hipdnnSetActivationDescriptor(actiDesc, mode,
        HIPDNN_NOT_PROPAGATE_NAN, 0.0));
  }
  // allocate tensors
  size_t outputSize = sizeof(DATATYPE) * n * c * h * w;
  size_t biasSize = sizeof(DATATYPE) * outputC;
  checkCUDA(hipMalloc(&biasPtr, biasSize));
  checkCUDA(hipMalloc(&outputs[0].data_ptr, outputSize));
}

void Conv2D::unmap(void)
{
  checkCUDNN(hipdnnDestroyTensorDescriptor(inputTensor));
  checkCUDNN(hipdnnDestroyTensorDescriptor(biasTensor));
  checkCUDNN(hipdnnDestroyTensorDescriptor(outputTensor));
  checkCUDNN(hipdnnDestroyFilterDescriptor(filterDesc));
  checkCUDNN(hipdnnDestroyConvolutionDescriptor(convDesc));
  if (activation != AC_MODE_NONE) {
    checkCUDNN(hipdnnDestroyActivationDescriptor(actiDesc));
  }
  // free tensors
  checkCUDA(hipFree(outputs[0].data_ptr));
  checkCUDA(hipFree(biasPtr));
}

void Conv2D::forward(bool block)
{
  const float alpha = 1.0f;
  const float beta = 0.0f;
  if (activation != AC_MODE_NONE) {
    checkCUDNN(cudnnConvolutionBiasActivationForward(
        model->dnn, &alpha, inputTensor, inputs[0].data_ptr, filterDesc, inputs[1].data_ptr,
        convDesc, fwdAlgo, model->workSpace, model->workSpaceSize,
        &beta, outputTensor, outputs[0].data_ptr, biasTensor, biasPtr, actiDesc,
        outputTensor, outputs[0].data_ptr));
  } else {
    checkCUDNN(hipdnnConvolutionForward(
        model->dnn, &alpha, inputTensor, inputs[0].data_ptr, filterDesc, inputs[1].data_ptr,
        convDesc, fwdAlgo, model->workSpace, model->workSpaceSize,
        &beta, outputTensor, outputs[0].data_ptr));
    checkCUDNN(hipdnnAddTensor(model->dnn, &alpha, biasTensor, biasPtr,
        &alpha, outputTensor, outputs[0].data_ptr));
  }
  if (block)
    checkCUDA(hipDeviceSynchronize());
}

void Model::measure_conv2d_cost(Conv2D* conv)
{
  const float alpha = 1.0f;
  const float beta = 0.0f;
  int inputN = conv->inputs[0].dim[0];
  int inputC = conv->inputs[0].dim[1];
  int inputH = conv->inputs[0].dim[2];
  int inputW = conv->inputs[0].dim[3];
  int kernelH = conv->inputs[1].dim[2];
  int kernelW = conv->inputs[1].dim[3];
  int outputC = conv->outputs[0].dim[1];
  int outputH = conv->outputs[0].dim[2];
  int outputW = conv->outputs[0].dim[3];
  int groups = conv->inputs[0].dim[1] / conv->inputs[1].dim[1];
  int padH, padW;
  // Reference: https://www.tensorflow.org/api_guides/python/nn#Convolution
  switch (conv->padding) {
    case PD_MODE_SAME:
      int totalPadH, totalPadW;
      if (inputH % conv->strideH == 0)
        totalPadH = max(kernelH - conv->strideH, 0);
      else
        totalPadH = max(kernelH - (inputH % conv->strideH), 0);
      if (inputW % conv->strideW == 0)
        totalPadW = max(kernelW - conv->strideW, 0);
      else
        totalPadW = max(kernelW - (inputW % conv->strideW), 0);
      // assert same padding on both sides
      padH = (totalPadH + 1) / 2;
      padW = (totalPadW + 1) / 2;
      break;
    case PD_MODE_VALID:
      padH = 0;
      padW = 0;
      break;
    default:
      assert(false);
  }
  checkCUDNN(hipdnnSetTensor4dDescriptor(inputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, inputN, inputC, inputH, inputW));
  checkCUDNN(hipdnnSetTensor4dDescriptor(biasTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, 1, outputC, 1, 1));
  checkCUDNN(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT,
      HIPDNN_TENSOR_NCHW, conv->inputs[1].dim[0], conv->inputs[1].dim[1],
      conv->inputs[1].dim[2], conv->inputs[1].dim[3]));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc, padH, padW,
      conv->strideH, conv->strideW, 1/*dilationH*/, 1/*dilationW*/,
      HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
  checkCUDNN(hipdnnSetConvolutionMathType(convDesc, HIPDNN_TENSOR_OP_MATH));
  checkCUDNN(hipdnnSetConvolutionGroupCount(convDesc, groups));
  checkCUDNN(hipdnnSetActivationDescriptor(actiDesc, HIPDNN_ACTIVATION_RELU,
        HIPDNN_NOT_PROPAGATE_NAN, 0.0));
  int n, c, h, w;
  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc,
      inputTensor, filterDesc, &n, &c, &h, &w));
  assert(n == inputN);
  assert(c == outputC);
  assert(outputH == h);
  assert(outputW == w);
  checkCUDNN(hipdnnSetTensor4dDescriptor(outputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, n, c, h, w));
  size_t inputSize = sizeof(DATATYPE) * inputN * inputC * inputH * inputW;
  size_t filterSize = sizeof(DATATYPE) * inputC * outputC
                      * kernelH * kernelW;
  size_t outputSize = sizeof(DATATYPE) * n * c * h * w;
  assert(inputSize < MAX_TENSOR_SIZE);
  assert(filterSize < MAX_TENSOR_SIZE);
  assert(outputSize < MAX_TENSOR_SIZE);

  const int reqAlgCnt = 8;
  int cnt = 0;
  hipdnnConvolutionFwdAlgoPerf_t perfResults[reqAlgCnt];
  checkCUDNN(hipdnnFindConvolutionForwardAlgorithmEx(
      dnn, inputTensor, inputPtr, filterDesc, filterPtr, convDesc,
      outputTensor, outputPtr, reqAlgCnt, &cnt, perfResults,
      workSpace, workSpaceSize));
  assert(cnt > 0);
  checkCUDNN(perfResults[0].status);
  //for (int i = 0; i < cnt; i++) {
    //printf("fwdAlgo(%d) time(%.2lfms) space(%dMB)\n", perfResults[i].algo,
    //       perfResults[i].time, perfResults[i].memory / 1024 / 1024);
  //}
  conv->fwdAlgo = perfResults[0].algo;
 
  checkCUDA(hipDeviceSynchronize());
  for (int i = 0; i < WARMUP_TIMES + REPEAT_TIMES; i++) {
    if (i == WARMUP_TIMES) {
      checkCUDA(hipEventRecord(startEvent));
    }
    if (conv->activation != AC_MODE_NONE) {
      checkCUDNN(cudnnConvolutionBiasActivationForward(
          dnn, &alpha, inputTensor, inputPtr, filterDesc, filterPtr,
          convDesc, conv->fwdAlgo, workSpace, workSpaceSize,
          &beta, outputTensor, outputPtr, biasTensor, biasPtr, actiDesc,
          outputTensor, outputPtr));
    } else {
      checkCUDNN(hipdnnConvolutionForward(
          dnn, &alpha, inputTensor, inputPtr, filterDesc, filterPtr,
          convDesc, conv->fwdAlgo, workSpace, workSpaceSize,
          &beta, outputTensor, outputPtr));
      checkCUDNN(hipdnnAddTensor(dnn, &alpha, biasTensor, biasPtr,
          &alpha, outputTensor, outputPtr));
    }
  }
  checkCUDA(hipEventRecord(endEvent));
  checkCUDA(hipEventSynchronize(endEvent));
  float milliseconds;
  hipEventElapsedTime(&milliseconds, startEvent, endEvent);
  conv->runtime = milliseconds / REPEAT_TIMES;
  if (print_cost)
    printf("  measure[Conv2D]: i(%d %d %d %d) w(%d %d %d %d) s(%d %d) p(%d %d) cost(%.4lf)\n",
           conv->inputs[0].dim[0], conv->inputs[0].dim[1], conv->inputs[0].dim[2], conv->inputs[0].dim[3],
           conv->inputs[1].dim[0], conv->inputs[1].dim[1], conv->inputs[1].dim[2], conv->inputs[1].dim[3],
           conv->strideH, conv->strideW, padH, padW, conv->runtime);
}

