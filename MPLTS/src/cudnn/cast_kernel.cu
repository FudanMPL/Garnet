/* Copyright 2019 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "taso/ops.h"
#include "taso/cuda_helper.h"
using namespace taso;

void Cast::map(void)
{
  checkCUDA(hipMalloc(&outputs[0].data_ptr, outputs[0].volume() * sizeof(DATATYPE)));
}

void Cast::unmap(void)
{
  checkCUDA(hipFree(outputs[0].data_ptr));
}

void Cast::forward(bool block)
{
  if (block)
    checkCUDA(hipDeviceSynchronize());
}

void Model::measure_cast_cost(Cast* cast)
{
  cast->runtime = 0;
  if (print_cost)
    printf("  measure[Cast]: type(%d) cost(%.4lf)\n",
           cast->type, cast->runtime);
}

