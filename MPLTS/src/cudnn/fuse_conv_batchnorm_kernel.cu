#include "hip/hip_runtime.h"
/* Copyright 2020 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "taso/ops.h"
#include "taso/cuda_helper.h"
using namespace taso;

__global__
void fuse_conv_batchnorm_kernel(int c_out,
                                int c_in_h_w,
                                DATATYPE* dst_ptr,
                                DATATYPE* conv_w,
                                DATATYPE* scale,
                                DATATYPE* var)
{
  int volume = c_out * c_in_h_w;
  CUDA_KERNEL_LOOP(i, volume)
  {
    int c_out_idx = i / c_in_h_w;
    dst_ptr[i] = scale[c_out_idx] * conv_w[i] / sqrt(abs(var[c_out_idx]) + HIPDNN_BN_MIN_EPSILON);
  }
}

void FuseConvBatchNorm::map(void)
{
  assert(inputs[0].numDim == 4);
  size_t outputSize = sizeof(DATATYPE) * outputs[0].volume();
  checkCUDA(hipMalloc(&outputs[0].data_ptr, outputSize));
}

void FuseConvBatchNorm::unmap(void)
{
  checkCUDA(hipFree(outputs[0].data_ptr));
}

void FuseConvBatchNorm::forward(bool block)
{
  int c_out = outputs[0].dim[0];
  int c_in_h_w = outputs[0].volume() / c_out;
  DATATYPE* conv_w_ptr = (DATATYPE*) inputs[0].data_ptr;
  DATATYPE* scale_ptr = (DATATYPE*) inputs[1].data_ptr;
  DATATYPE* var_ptr = (DATATYPE*) inputs[4].data_ptr;
  fuse_conv_batchnorm_kernel<<<GET_BLOCKS(outputs[0].volume()), CUDA_NUM_THREADS>>>(
      c_out, c_in_h_w, (DATATYPE*)outputs[0].data_ptr,
      conv_w_ptr, scale_ptr, var_ptr);
  if (block)
    checkCUDA(hipDeviceSynchronize());
}


