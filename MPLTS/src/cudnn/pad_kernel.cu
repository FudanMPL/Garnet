/* Copyright 2019 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "taso/ops.h"
#include "taso/cuda_helper.h"
using namespace taso;

void Pad::map(void)
{
  checkCUDA(hipMalloc(&outputs[0].data_ptr, outputs[0].volume() * sizeof(DATATYPE)));
}

void Pad::unmap(void)
{
  checkCUDA(hipFree(outputs[0].data_ptr));
}

void Pad::forward(bool block)
{
  if (block)
    checkCUDA(hipDeviceSynchronize());
}

void Model::measure_pad_cost(Pad* pad)
{
  pad->runtime = 0;
}
