/* Copyright 2019 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "taso/ops.h"
#include "taso/cuda_helper.h"
using namespace taso;

void Transpose::map(void)
{
  //TODO: for now the output and input share the same instance
  outputs[0].data_ptr = inputs[0].data_ptr;
}

void Transpose::unmap(void)
{
}

void Transpose::forward(bool block)
{
  if (block)
    checkCUDA(hipDeviceSynchronize());
}

void Model::measure_transpose_cost(Transpose* transpose)
{
  // Transpose requires no kernel launch
  transpose->runtime = 0;
}
