/* Copyright 2019 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "taso/ops.h"
#include "taso/cuda_helper.h"
using namespace taso;

void Where::map(void)
{
  checkCUDA(hipMalloc(&outputs[0].data_ptr, outputs[0].volume() * sizeof(DATATYPE)));
}

void Where::unmap(void)
{
  checkCUDA(hipFree(outputs[0].data_ptr));
}

void Where::forward(bool block)
{
  if (block)
    checkCUDA(hipDeviceSynchronize());
}

void Model::measure_where_cost(Where* where)
{
  where->runtime = 0;
}
