#include "aes.cu"
#include "utils.cu"
#include "fss_struct.h"
#include "fss.cu"

void test_add(uint8_t * a, uint8_t * b, uint8_t * res, int numbytes){
  uint8_t *buf_a;
  uint8_t *buf_b;
  uint8_t *buf_res;

  hipMalloc((void**)&buf_a, numbytes);
  hipMalloc((void**)&buf_b, numbytes);
  hipMalloc((void**)&buf_res, numbytes);
  //从内存拷贝至显存
  hipMemcpy(buf_a, a, numbytes, hipMemcpyHostToDevice);
  hipMemcpy(buf_b, b, numbytes, hipMemcpyHostToDevice);

  _add<<<1, 1>>>(buf_a, buf_b, buf_res, numbytes);
  hipMemcpy(res, buf_res, numbytes, hipMemcpyDeviceToHost);
  return;
}

void test_restricted_multiply(int value, uint8_t * a, uint8_t * res, int numbytes){
  uint8_t *buf_a;
  uint8_t *buf_res;

  hipMalloc((void**)&buf_a, numbytes);
  hipMalloc((void**)&buf_res, numbytes);
  //从内存拷贝至显存
  hipMemcpy(buf_a, a, numbytes, hipMemcpyHostToDevice);

  _restricted_multiply<<<1,numbytes>>>(value, buf_a,  buf_res, numbytes);
  hipMemcpy(res, buf_res, numbytes, hipMemcpyDeviceToHost);
  return;
}

void test_xor(uint8_t * a, uint8_t * b, uint8_t * res, int numbytes){
  uint8_t *buf_a;
  uint8_t *buf_b;
  uint8_t *buf_res;

  hipMalloc((void**)&buf_a, numbytes);
  hipMalloc((void**)&buf_b, numbytes);
  hipMalloc((void**)&buf_res, numbytes);
  //从内存拷贝至显存
  hipMemcpy(buf_a, a, numbytes, hipMemcpyHostToDevice);
  hipMemcpy(buf_b, b, numbytes, hipMemcpyHostToDevice);

  _xor<<<1, numbytes>>>(buf_a, buf_b, buf_res, numbytes);
  hipMemcpy(res, buf_res, numbytes, hipMemcpyDeviceToHost);
  return;
}

void test_sub(uint8_t * a, uint8_t * b, uint8_t * res, int numbytes){
  uint8_t *buf_a;
  uint8_t *buf_b;
  uint8_t *buf_res;

  hipMalloc((void**)&buf_a, numbytes);
  hipMalloc((void**)&buf_b, numbytes);
  hipMalloc((void**)&buf_res, numbytes);
  //从内存拷贝至显存
  hipMemcpy(buf_a, a, numbytes, hipMemcpyHostToDevice);
  hipMemcpy(buf_b, b, numbytes, hipMemcpyHostToDevice);

  _sub<<<1, 1>>>(buf_a, buf_b, buf_res, numbytes);
  hipMemcpy(res, buf_res, numbytes, hipMemcpyDeviceToHost);
  return;
}

//uint8_t * seed_0, uint8_t * seed_1分别是初始化后的随机数种子
//uint8_t * generated_value_cpu_0是表示给party0生成的随机数结果存放位置， uint8_t * generated_value_cpu_1是表示给party1生成的随机数结果存放位置
void fss_generate(uint8_t * r, uint8_t * seed0, uint8_t * seed1, uint8_t * generated_value_cpu, int numbytes, int parallel){
  uint8_t * generated_value;
  
  int block_number = parallel;
  int bit_length = numbytes * 8;
  
  //分配扩展密钥
  BYTE key[16 * (14 + 1)];
  int keyLen = 16;
  int blockLen = 16;

  hipSetDevice(0);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  int num_sm = prop.multiProcessorCount; 
  BYTE *cuda_key;//, *cuda_Sbox;
  int expandKeyLen = AES_ExpandKey(key, keyLen);
  int thrdperblock = block_number/num_sm;
  for(int i = 0; i < blockLen; i++){
      key[i] = i;
  }
  AES_ExpandKey(key, keyLen);
  hipMalloc(&cuda_key,16*15*sizeof(BYTE) );
  hipMemcpy(cuda_key, key, 16*15*sizeof(BYTE), hipMemcpyHostToDevice); 
  
  
  // for (int i = 0; i < blockLen; i++){
  //     key[i] = key[i] * 2;
  // }
  // AES_ExpandKey(key, keyLen);
  // hipMalloc(&cuda_key[1],16*15*sizeof(BYTE) );
  // hipMemcpy(cuda_key[1], key, 16*15*sizeof(BYTE), hipMemcpyHostToDevice); 

  if(block_number%num_sm>0)
      thrdperblock++;

  if(thrdperblock>1024){
      thrdperblock = 1024;
      num_sm = block_number/1024;
      if(block_number%1024>0){
          num_sm++;
      }
  }
  dim3 ThreadperBlock(thrdperblock);
  dim3 BlockperGrid(num_sm);
  
  // //分配输出的generated_value的长度
  hipMalloc((void**)&generated_value, (bit_length - 1) * 2 * (numbytes + 1) + numbytes);
  //分配数据结构空间
  
  FssGen * fss_gen = new FssGen();
  FssGen * cuda_fss_gen;
  for(int i = 0; i < 2; i++){
    fss_gen->pre_t[i] = i;
  }
  
  hipMalloc(&cuda_fss_gen, sizeof(class FssGen));
  hipMemcpy(&cuda_fss_gen->pre_t, fss_gen->pre_t, 2*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(&cuda_fss_gen->seed[0], seed0, numbytes * sizeof(BYTE), hipMemcpyHostToDevice);
  hipMemcpy(&cuda_fss_gen->seed[1], seed1, numbytes * sizeof(BYTE), hipMemcpyHostToDevice);
  
  //记录加密算法开始时间
  hipEvent_t start1;
  hipEventCreate(&start1);
  hipEvent_t stop1;
  hipEventCreate(&stop1);
  hipEventRecord(start1);

  int keep, lose;
  for(int i = 0; i < bit_length-1; i++){
    int idx = int(i/8);
    keep = ((r[idx]) >> (7 - (i - (idx) * 8)))%2;
    lose = keep ^ 1;       
    
    for(int j = 0; j < 2; j++){
      _copy<<<1,numbytes>>>(cuda_fss_gen->seed[j], cuda_fss_gen->inter_val[j], 0, 0, numbytes);
      _copy<<<1,numbytes>>>(cuda_fss_gen->seed[j], cuda_fss_gen->inter_val[j], 0, numbytes, numbytes);
      _copy<<<1,numbytes>>>(cuda_fss_gen->seed[j], cuda_fss_gen->inter_val[j], 0, 2*numbytes, 1);
      
      // printGpuBytes<<<1,1>>>(cuda_fss_gen->inter_val[j], 2*numbytes+1);
      
      for(int k = 0; k < 2; k++){
        AES_Encrypt<<<num_sm, thrdperblock>>>(cuda_fss_gen->inter_val[j], cuda_key, 176, numbytes, 3);
        _copy<<<1,numbytes>>>(cuda_fss_gen-> inter_val[j], cuda_fss_gen->t[j][0], 0, 0, 1);
        _mod2_t<<<1,1>>>(cuda_fss_gen->t[j][0]);
        _copy<<<1,numbytes>>>(cuda_fss_gen->inter_val[j], cuda_fss_gen->v[j][0], 1, 0, numbytes);
        _copy<<<1,numbytes>>>(cuda_fss_gen->inter_val[j], cuda_fss_gen->s[j][0], numbytes + 1, 0,  numbytes);

        AES_Encrypt<<<num_sm, thrdperblock>>>(cuda_fss_gen->inter_val[j], cuda_key, 176, numbytes, 3);
        _copy<<<1,numbytes>>>(cuda_fss_gen-> inter_val[j], cuda_fss_gen->t[j][1],  0, 0, 1);
        _mod2_t<<<1,1>>>(cuda_fss_gen->t[j][1]);
        _copy<<<1,numbytes>>>(cuda_fss_gen->inter_val[j], cuda_fss_gen->v[j][1], 1, 0, numbytes);
        _copy<<<1,numbytes>>>(cuda_fss_gen->inter_val[j], cuda_fss_gen->s[j][1], 1 + numbytes, 0, numbytes);
      }
    }
    _xor<<<1,numbytes>>>(cuda_fss_gen->s[lose][0], cuda_fss_gen->s[lose][1], cuda_fss_gen->scw, numbytes);
    for(int j = 0; j < 2; j++){
        _copy<<<1,numbytes>>>(cuda_fss_gen->v[lose][j], cuda_fss_gen->convert_seed[j], 0, 0, numbytes);
        AES_Encrypt<<<num_sm, thrdperblock>>>(cuda_fss_gen->seed[j], cuda_key, 176, numbytes, 1);
        _copy<<<1,numbytes>>>(cuda_fss_gen->convert_seed[j], cuda_fss_gen->convert[i], 0, 0, numbytes);
    }
    if(keep){
      vcw_generate_update_keep<<<1,1>>>(cuda_fss_gen, numbytes);
    }
    else{
      vcw_generate_update_lose<<<1,1>>>(cuda_fss_gen, numbytes);
    }
    for(int j = 0; j < 2; j++){
        _copy<<<1,numbytes>>>(cuda_fss_gen->v[keep][j], cuda_fss_gen->convert_seed[j], 0, 0, numbytes);
        AES_Encrypt<<<num_sm, thrdperblock>>>(cuda_fss_gen->convert_seed[j], cuda_key, 176, numbytes, 1);
        _copy<<<1,numbytes>>>(cuda_fss_gen->convert_seed[j], cuda_fss_gen->convert[j], 0, 0, numbytes);
    }
    va_genearte_update<<<1,1>>>(cuda_fss_gen, keep, numbytes);
    tcw_pret_generate_update<<<1,1>>>(cuda_fss_gen, keep, numbytes);
    _copy<<<1,numbytes>>>(cuda_fss_gen->scw, generated_value, 0, 2*(numbytes+1)*i, numbytes);
    _copy<<<1,numbytes>>>(cuda_fss_gen->vcw, generated_value, 0, 2*(numbytes+1)*i + numbytes, numbytes); 
    _copy<<<1,numbytes>>>(cuda_fss_gen->tcw[0], generated_value, 0, 2*(numbytes+1)*i + 2 * numbytes, 1);
    _copy<<<1,numbytes>>>(cuda_fss_gen->tcw[1], generated_value, 0, 2*(numbytes+1)*i + 2 * numbytes + 1, 1);
  }
  
  for(int j = 0; j < 2; j++){
      _copy<<<1,numbytes>>>(cuda_fss_gen->seed[j], cuda_fss_gen->convert_seed[j], 0, 0, numbytes);
      AES_Encrypt<<<num_sm, thrdperblock>>>(cuda_fss_gen->convert_seed[j], cuda_key, 176, numbytes, 1);
      _copy<<<1,numbytes>>>(cuda_fss_gen->convert_seed[j], cuda_fss_gen->convert[j], 0, 0, numbytes);
  }
  
  final_cw_generate_update<<<1,1>>>(cuda_fss_gen, generated_value, bit_length, numbytes);

  //记录加密算法结束时间，并计算加密速度
  hipEventRecord(stop1);
  hipEventSynchronize(stop1);
  float msecTotal1,total;
  hipEventElapsedTime(&msecTotal1, start1, stop1);
  total=msecTotal1/1000;
  printf("time:%f\n",total);


  hipMemcpy(generated_value_cpu, generated_value, (bit_length - 1) * 2 * (numbytes + 1) + numbytes, hipMemcpyDeviceToHost);

}


void fss_evaluate(int party, uint8_t * x_reveal, uint8_t * seed, uint8_t * gen_val, uint8_t * result, int numbytes, int parallel){
  int bit_length = numbytes * 8;
  uint8_t * generated_value;
  //分配扩展密钥
  BYTE key[16 * (14 + 1)];
  int keyLen = 16;
  int blockLen = 16;

  hipSetDevice(0);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  int num_sm = prop.multiProcessorCount; 
  BYTE *cuda_key;//, *cuda_Sbox;
  int expandKeyLen = AES_ExpandKey(key, keyLen);
  int thrdperblock = parallel/num_sm;
  for(int i = 0; i < blockLen; i++){
      key[i] = i;
  }
  AES_ExpandKey(key, keyLen);
  hipMalloc(&cuda_key,16*15*sizeof(BYTE) );
  hipMemcpy(cuda_key, key, 16*15*sizeof(BYTE), hipMemcpyHostToDevice); 
  
  if(parallel%num_sm>0)
      thrdperblock++;

  if(thrdperblock>1024){
      thrdperblock = 1024;
      num_sm = parallel/1024;
      if(parallel%1024>0){
          num_sm++;
      }
  }
  dim3 ThreadperBlock(thrdperblock);
  dim3 BlockperGrid(num_sm);
  
  // //分配输出的generated_value的长度
  hipMalloc((void**)&generated_value, (bit_length - 1) * 2 * (numbytes + 1) + numbytes);
  //分配数据结构空间
  
  FssEval * fss_eval = new FssEval();
  FssEval * cuda_fss_eval;
  fss_eval->pre_t = party;
  
  hipMalloc(&cuda_fss_eval, sizeof(class FssEval));
  hipMemcpy(&cuda_fss_eval, fss_eval, sizeof(class FssEval), hipMemcpyHostToDevice);
  hipMemcpy(&cuda_fss_eval->seed, seed, numbytes * sizeof(BYTE), hipMemcpyHostToDevice);

  //记录加密算法开始时间
  hipEvent_t start1;
  hipEventCreate(&start1);
  hipEvent_t stop1;
  hipEventCreate(&stop1);
  hipEventRecord(start1);
  for(int i = 0; i < bit_length-1; i++){
    int idx = int(i/8);
    int xi = ((x_reveal[idx]) >> (7 - (i - (idx) * 8)))%2;
    correction_word_eval_copy<<<1,1>>>(cuda_fss_eval, generated_value, numbytes, i);
    random_value_eval_generate<<<1,2>>>(cuda_fss_eval, cuda_key, numbytes, num_sm, thrdperblock);
    convert_random_value_eval_generate<<<1,2>>>(cuda_fss_eval, cuda_key, numbytes, num_sm, thrdperblock);
    value_eval_update<<<1,1>>>(cuda_fss_eval, xi, party, numbytes);
  }
  final_eval_update<<<1,1>>>(cuda_fss_eval, cuda_key, generated_value, party, numbytes, num_sm, thrdperblock);
  hipEventRecord(stop1);
  hipEventSynchronize(stop1);
  float msecTotal1,total;
  hipEventElapsedTime(&msecTotal1, start1, stop1);
  total=msecTotal1/1000;
  printf("eval time:%f\n",total);
  hipMemcpy(cuda_fss_eval->tmp_v, result, numbytes, hipMemcpyDeviceToHost);
}