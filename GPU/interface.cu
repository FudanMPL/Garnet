#include "aes.cu"
#include "utils.cu"
#include "fss_struct.h"
#include "fss.cu"
#include "test.cu"

void fss_dpf_generate(RandomValueBlock * cpu_r_block, aes_gen_block * cpu_aes_block_array, CorrectionWord * cpu_cw, int parallel){
    int lambda = 127;
    int bit_length = INPUT_BYTE * 8;

    
    BYTE key[240];
    int keyLen = 16;
    int blockLen = 16;
    KeyBlock * cuda_key_block;
    hipMalloc(&cuda_key_block, 2 * sizeof(class KeyBlock));
    prepare_key(cuda_key_block, key, keyLen);

    // printGpuBytes<<<1,1>>>(cuda_key_block->cuda_key[0], 0, 240);
    int thrdperblock, num_sm;
    init_sm_thrd(num_sm, thrdperblock, parallel);
    std::cout << num_sm << " " << thrdperblock <<  std::endl;

    dim3 ThreadperBlock(thrdperblock);
    dim3 BlockperGrid(num_sm);


    CorrectionWord * cuda_cw;   
    FssDpfGen * cuda_dpf_gen;   
    RandomValueBlock * cuda_r_block;   
    aes_gen_block * cuda_aes_block_array;
    hipMalloc(&cuda_cw, parallel*sizeof(class CorrectionWord));
    hipMalloc(&cuda_dpf_gen, parallel*sizeof(class FssDpfGen));
    hipMalloc(&cuda_r_block, parallel*sizeof(class RandomValueBlock));
    hipMemcpy(cuda_r_block, cpu_r_block, parallel*sizeof(class RandomValueBlock), hipMemcpyHostToDevice);
    hipMalloc(&cuda_aes_block_array, parallel*sizeof(class aes_gen_block));
    hipMemcpy(cuda_aes_block_array, cpu_aes_block_array, parallel*sizeof(class aes_gen_block), hipMemcpyHostToDevice);
    
    hipDeviceSynchronize();
    hipEvent_t start1;
    hipEventCreate(&start1);
    hipEvent_t stop1;
    hipEventCreate(&stop1);
    hipEventRecord(start1);

    gen_init<<<BlockperGrid, ThreadperBlock>>>(cuda_dpf_gen, parallel);
    for(int i = 0; i < bit_length; i++){        
        for(int j = 0; j < 2; j++){
            AES_Encrypt_Gen<<<BlockperGrid,ThreadperBlock>>>(cuda_aes_block_array, cuda_key_block, 176, j, parallel);
            st_copy_gen<<<BlockperGrid,ThreadperBlock>>>(cuda_aes_block_array, cuda_dpf_gen, j, parallel); 
        }
        cw_update_gen<<<BlockperGrid, ThreadperBlock>>>(cuda_r_block, cuda_cw, cuda_dpf_gen, i, parallel);        
        
        for(int b = 0; b < 2; b++){
            st_update_gen<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_cw, cuda_dpf_gen, i, b, parallel);
        }
    }
    final_cw_update_gen<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_cw, cuda_dpf_gen, parallel);

    hipMemcpy(cpu_cw, cuda_cw, parallel*sizeof(class CorrectionWord), hipMemcpyDeviceToHost);

    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    float msecTotal1,total;
    hipEventElapsedTime(&msecTotal1, start1, stop1);
    total=msecTotal1/1000;
    printf("eval time:%f\n",total);
    hipFree(cuda_dpf_gen);
    hipFree(cuda_key_block);
    hipFree(cuda_cw);
    hipFree(cuda_r_block);
    hipFree(cuda_aes_block_array);
}


void fss_dpf_evaluate(RevealValueBlock * cpu_reveal, aes_eval_block * cpu_aes_block_array, CorrectionWord * cpu_cw, ResultBlock * cpu_res, bool party, int parallel){
    int lambda = 127;
    int bit_length = INPUT_BYTE * 8;

    
    BYTE key[240];
    int keyLen = 16;
    int blockLen = 16;
    KeyBlock * cuda_key_block;
    hipMalloc(&cuda_key_block, 2 * sizeof(class KeyBlock));
    prepare_key(cuda_key_block, key, keyLen); 

    int thrdperblock, num_sm;
    init_sm_thrd(num_sm, thrdperblock, parallel);
    std::cout << num_sm << " " << thrdperblock <<  std::endl;

    dim3 ThreadperBlock(thrdperblock);
    dim3 BlockperGrid(num_sm);

    CorrectionWord * cuda_cw;   
    FssDpfEval * cuda_dpf_eval;   
    RevealValueBlock * cuda_reveal_block;   
    aes_eval_block * cuda_aes_block_array;
    ResultBlock * cuda_res;
    hipMalloc(&cuda_cw, parallel*sizeof(class CorrectionWord));
    hipMemcpy(cuda_cw, cpu_cw, parallel*sizeof(class CorrectionWord), hipMemcpyHostToDevice);
    hipMalloc(&cuda_dpf_eval, parallel*sizeof(class FssDpfEval));
    hipMalloc(&cuda_reveal_block, parallel*sizeof(class RevealValueBlock));
    hipMemcpy(cuda_reveal_block, cpu_reveal, parallel*sizeof(class RevealValueBlock), hipMemcpyHostToDevice);
    hipMalloc(&cuda_aes_block_array, parallel*sizeof(class aes_eval_block));
    hipMemcpy(cuda_aes_block_array, cpu_aes_block_array, parallel*sizeof(class aes_eval_block), hipMemcpyHostToDevice);
    hipMalloc(&cuda_res, parallel*sizeof(class ResultBlock));

    hipDeviceSynchronize();
    hipEvent_t start1;
    hipEventCreate(&start1);
    hipEvent_t stop1;
    hipEventCreate(&stop1);
    hipEventRecord(start1);
    eval_init<<<BlockperGrid, ThreadperBlock>>>(cuda_dpf_eval, party, parallel);
    for(int i = 0; i < bit_length; i++){
        AES_Encrypt_Eval<<<BlockperGrid,ThreadperBlock>>>(cuda_aes_block_array, cuda_key_block, 176, parallel);
        test<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_dpf_eval, parallel);
        st_init_eval<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_cw, cuda_dpf_eval, i, parallel);
        st_update_eval<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_reveal_block, cuda_cw, cuda_dpf_eval, i, parallel);  
    }
    result_update_eval<<<BlockperGrid, ThreadperBlock>>>(cuda_res, cuda_aes_block_array, cuda_cw, cuda_dpf_eval, parallel);
    hipMemcpy(cpu_res, cuda_res, parallel*sizeof(class ResultBlock), hipMemcpyDeviceToHost);
    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    float msecTotal1,total;
    hipEventElapsedTime(&msecTotal1, start1, stop1);
    total=msecTotal1/1000;
    printf("eval time:%f\n",total);
    hipFree(cuda_dpf_eval);
    hipFree(cuda_key_block);
    hipFree(cuda_cw);
    hipFree(cuda_reveal_block);
    hipFree(cuda_aes_block_array);
}