#include "hip/hip_runtime.h"
#include "aes.cu"
#include "utils.cu"
#include "fss_struct.h"
#include "fss.cu"
#include "test.cu"

void fss_dpf_generate(RandomValueBlock * cpu_r_block, aes_gen_block * cpu_aes_block_array, CorrectionWord * cpu_cw, int parallel){
    int lambda = 127;
    int bit_length = INPUT_BYTE * 8;

    
    BYTE key[240];
    int keyLen = 16;
    int blockLen = 16;
    KeyBlock * cuda_key_block;
    hipMalloc(&cuda_key_block, 2 * sizeof(class KeyBlock));
    prepare_key(cuda_key_block, key, keyLen);

    // printGpuBytes<<<1,1>>>(cuda_key_block->cuda_key[0], 0, 240);
    int thrdperblock, num_sm;
    init_sm_thrd(num_sm, thrdperblock, parallel);
    std::cout << num_sm << " " << thrdperblock <<  std::endl;

    dim3 ThreadperBlock(thrdperblock);
    dim3 BlockperGrid(num_sm);


    CorrectionWord * cuda_cw;   
    FssDpfGen * cuda_dpf_gen;   
    RandomValueBlock * cuda_r_block;   
    aes_gen_block * cuda_aes_block_array;
    hipMalloc(&cuda_cw, parallel*sizeof(class CorrectionWord));
    hipMalloc(&cuda_dpf_gen, parallel*sizeof(class FssDpfGen));
    hipMalloc(&cuda_r_block, parallel*sizeof(class RandomValueBlock));
    hipMemcpy(cuda_r_block, cpu_r_block, parallel*sizeof(class RandomValueBlock), hipMemcpyHostToDevice);
    hipMalloc(&cuda_aes_block_array, parallel*sizeof(class aes_gen_block));
    hipMemcpy(cuda_aes_block_array, cpu_aes_block_array, parallel*sizeof(class aes_gen_block), hipMemcpyHostToDevice);
    
    hipDeviceSynchronize();
   
    // std::cout << "copy finished!" << std::endl;
    //记录加密算法开始时间
    hipEvent_t start1;
    hipEventCreate(&start1);
    hipEvent_t stop1;
    hipEventCreate(&stop1);
    hipEventRecord(start1);

    // int count;
 
    // hipGetDeviceCount(&count);
    // printf("gpu num %d\n", count);
    // hipGetDeviceProperties(&prop, 0);
    // printf("max thread num: %d\n", prop.maxThreadsPerBlock);
    // printf("max grid dimensions: %d, %d, %d)\n",
    // prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    gen_init<<<BlockperGrid, ThreadperBlock>>>(cuda_dpf_gen, parallel);
    for(int i = 0; i < bit_length; i++){
        
        
        // printGpuBytes<<<1,1>>>(cuda_aes_block_array[0].block[0], 0, 2*LAMBDA_BYTE);
        printGpuBytes<<<1,1>>>(cuda_aes_block_array[0].block[1], 0, 2*LAMBDA_BYTE);
        for(int j = 0; j < 2; j++){
            AES_Encrypt_Gen<<<BlockperGrid,ThreadperBlock>>>(cuda_aes_block_array, cuda_key_block, 176, j, parallel);
            st_copy_gen<<<BlockperGrid,ThreadperBlock>>>(cuda_aes_block_array, cuda_dpf_gen, j, parallel); 
        }
        // printGpuBytes<<<1,1>>>(cuda_dpf_gen[0].s[0][0], 0, LAMBDA_BYTE);
        printGpuBytes<<<1,1>>>(cuda_dpf_gen[0].s[0][1], 0, LAMBDA_BYTE);
        // printGpuBytes<<<1,1>>>(cuda_dpf_gen[0].s[1][0], 0, LAMBDA_BYTE);
        printGpuBytes<<<1,1>>>(cuda_dpf_gen[0].s[1][1], 0, LAMBDA_BYTE);
        cw_update_gen<<<BlockperGrid, ThreadperBlock>>>(cuda_r_block, cuda_cw, cuda_dpf_gen, i, parallel);        
        
        for(int b = 0; b < 2; b++){
            st_update_gen<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_cw, cuda_dpf_gen, i, b, parallel);
        }
        // test<<<1,1>>>(cuda_dpf_gen, cuda_cw, 0, i);
        // aes_block_copy_gen<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, parallel);
    }

    final_cw_update_gen<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_cw, cuda_dpf_gen, parallel);

    hipMemcpy(cpu_cw, cuda_cw, parallel*sizeof(class CorrectionWord), hipMemcpyDeviceToHost);

    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    float msecTotal1,total;
    hipEventElapsedTime(&msecTotal1, start1, stop1);
    total=msecTotal1/1000;
    printf("eval time:%f\n",total);
    hipFree(cuda_dpf_gen);
    hipFree(cuda_key_block);
    hipFree(cuda_cw);
    hipFree(cuda_r_block);
    hipFree(cuda_aes_block_array);
}


void fss_dpf_evaluate(RevealValueBlock * cpu_reveal, aes_eval_block * cpu_aes_block_array, CorrectionWord * cpu_cw, ResultBlock * cpu_res, bool party, int parallel){
    int lambda = 127;
    int bit_length = INPUT_BYTE * 8;

    
    BYTE key[240];
    int keyLen = 16;
    int blockLen = 16;
    KeyBlock * cuda_key_block;
    hipMalloc(&cuda_key_block, 2 * sizeof(class KeyBlock));
    prepare_key(cuda_key_block, key, keyLen); 

    int thrdperblock, num_sm;
    init_sm_thrd(num_sm, thrdperblock, parallel);
    std::cout << num_sm << " " << thrdperblock <<  std::endl;

    dim3 ThreadperBlock(thrdperblock);
    dim3 BlockperGrid(num_sm);

    CorrectionWord * cuda_cw;   
    FssDpfEval * cuda_dpf_eval;   
    RevealValueBlock * cuda_reveal_block;   
    aes_eval_block * cuda_aes_block_array;
    ResultBlock * cuda_res;
    hipMalloc(&cuda_cw, parallel*sizeof(class CorrectionWord));
    hipMemcpy(cuda_cw, cpu_cw, parallel*sizeof(class CorrectionWord), hipMemcpyHostToDevice);
    hipMalloc(&cuda_dpf_eval, parallel*sizeof(class FssDpfEval));
    hipMalloc(&cuda_reveal_block, parallel*sizeof(class RevealValueBlock));
    hipMemcpy(cuda_reveal_block, cpu_reveal, parallel*sizeof(class RevealValueBlock), hipMemcpyHostToDevice);
    hipMalloc(&cuda_aes_block_array, parallel*sizeof(class aes_eval_block));
    hipMemcpy(cuda_aes_block_array, cpu_aes_block_array, parallel*sizeof(class aes_eval_block), hipMemcpyHostToDevice);
    hipMalloc(&cuda_res, parallel*sizeof(class ResultBlock));

    hipDeviceSynchronize();
    hipEvent_t start1;
    hipEventCreate(&start1);
    hipEvent_t stop1;
    hipEventCreate(&stop1);
    hipEventRecord(start1);
    eval_init<<<BlockperGrid, ThreadperBlock>>>(cuda_dpf_eval, party, parallel);
    for(int i = 0; i < bit_length; i++){
        printGpuBytes<<<1,1>>>(cuda_aes_block_array[0].block, 0, 2*LAMBDA_BYTE);
        AES_Encrypt_Eval<<<BlockperGrid,ThreadperBlock>>>(cuda_aes_block_array, cuda_key_block, 176, parallel);
        test<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_dpf_eval, parallel);
        printGpuBytes<<<1,1>>>(cuda_dpf_eval[0].s[0], 0, LAMBDA_BYTE);
        printGpuBytes<<<1,1>>>(cuda_dpf_eval[0].s[1], 0, LAMBDA_BYTE);
        st_init_eval<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_cw, cuda_dpf_eval, i, parallel);
        printGpuBytes<<<1,1>>>(cuda_dpf_eval[0].s[0], 0, LAMBDA_BYTE);
        printGpuBytes<<<1,1>>>(cuda_dpf_eval[0].s[1], 0, LAMBDA_BYTE);
        // test<<<1,1>>>(cuda_dpf_eval, cuda_cw, i);
        st_update_eval<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_reveal_block, cuda_cw, cuda_dpf_eval, i, parallel);
        // test<<<1,1>>>(cuda_dpf_eval, cuda_cw, i);
    }
    result_update_eval<<<BlockperGrid, ThreadperBlock>>>(cuda_res, cuda_aes_block_array, cuda_cw, cuda_dpf_eval, parallel);
    printGpuBytes<<<1,1>>>(cuda_res[0].result, 0, INPUT_BYTE);
    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    float msecTotal1,total;
    hipEventElapsedTime(&msecTotal1, start1, stop1);
    total=msecTotal1/1000;
    printf("eval time:%f\n",total);
    hipFree(cuda_dpf_eval);
    hipFree(cuda_key_block);
    hipFree(cuda_cw);
    hipFree(cuda_reveal_block);
    hipFree(cuda_aes_block_array);
}