#include "hip/hip_runtime.h"
#include "aes.cu"
#include "utils.cu"
#include "fss_struct.h"
#include "fss.cu"

//aes加密
void encryptdemo(uint8_t *key, uint8_t *buf, unsigned long numbytes){
  uint8_t *buf_d;
  uint8_t *w_d;
  uint8_t *w;

  hipMemcpyToSymbol(HIP_SYMBOL(sbox), sbox, sizeof(uint8_t)*256);

  //为扩展后密钥分配内存空间
  w = (uint8_t*)malloc(240*sizeof(uint8_t));
  
  aes_key_expansion(key, w);

  //为数据和扩展后的密钥分配显存空间
  hipMalloc((void**)&buf_d, numbytes);
  hipMalloc((void**)&w_d, 240*sizeof(uint8_t));
  //从内存拷贝至显存
  hipMemcpy(buf_d, buf, numbytes, hipMemcpyHostToDevice);
  hipMemcpy(w_d, w, 240*sizeof(uint8_t), hipMemcpyHostToDevice);

  //计算GRIDSIZE与BLOCKSIZE
  dim3 dimBlock(ceil((double)numbytes / (double)(THREADS_PER_BLOCK * AES_BLOCK_SIZE)));
  dim3 dimGrid(THREADS_PER_BLOCK);
  //对每个数据块进行aes加密
  aes256_encrypt_ecb<<<dimBlock, dimGrid>>>(buf_d, numbytes, w_d);

  hipMemcpy(buf, buf_d, numbytes, hipMemcpyDeviceToHost);
  
}

// aes解密
void decryptdemo(uint8_t *key, uint8_t *buf, unsigned long numbytes){
  uint8_t *buf_d;
  
  uint8_t *w;

  hipMemcpyToSymbol(HIP_SYMBOL(sboxinv), sboxinv, sizeof(uint8_t)*256);

  printf("\nBeginning decryption\n");

  //记录解密算法开始时间
  hipEvent_t start1;
  hipEventCreate(&start1);
  hipEvent_t stop1;
  hipEventCreate(&stop1);
  hipEventRecord(start1);

  //为扩展后密钥分配内存空间
  w = (uint8_t*)malloc(240*sizeof(uint8_t));

  aes_key_expansion(key, w);

  //分配显存空间
  hipMalloc((void**)&buf_d, numbytes);
  //从内存拷贝至显存
  hipMemcpy(buf_d, buf, numbytes, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(w3), w, 240*sizeof(uint8_t));

  //计算GRIDSIZE与BLOCKSIZE
  dim3 dimBlock(ceil((double)numbytes / (double)(THREADS_PER_BLOCK * AES_BLOCK_SIZE)));
  dim3 dimGrid(THREADS_PER_BLOCK);
  printf("Creating %d threads over %d blocks\n", dimBlock.x*dimGrid.x, dimBlock.x);
    //对每个数据块进行aes解密
  aes256_decrypt_ecb<<<dimBlock, dimGrid>>>(buf_d, numbytes);

  hipMemcpy(buf, buf_d, numbytes, hipMemcpyDeviceToHost);

  //记录解密算法结束时间，并计算解密速度
  hipEventRecord(stop1);
  hipEventSynchronize(stop1);
  float msecTotal1,total;
  hipEventElapsedTime(&msecTotal1, start1, stop1);
  total=msecTotal1/1000;
  printf("time:%f\n",total);
  printf("Throughput: %fGbps\n", numbytes/total/1024/1024/1024*8);
}

void test_add(uint8_t * a, uint8_t * b, uint8_t * res, int numbytes){
  uint8_t *buf_a;
  uint8_t *buf_b;
  uint8_t *buf_res;

  hipMalloc((void**)&buf_a, numbytes);
  hipMalloc((void**)&buf_b, numbytes);
  hipMalloc((void**)&buf_res, numbytes);
  //从内存拷贝至显存
  hipMemcpy(buf_a, a, numbytes, hipMemcpyHostToDevice);
  hipMemcpy(buf_b, b, numbytes, hipMemcpyHostToDevice);

  _add<<<1, 1>>>(buf_a, buf_b, buf_res, numbytes);
  hipMemcpy(res, buf_res, numbytes, hipMemcpyDeviceToHost);
  return;
}

void test_restricted_multiply(int value, uint8_t * a, uint8_t * res, int numbytes){
  uint8_t *buf_a;
  uint8_t *buf_res;
  int *buf_value;

  hipMalloc((void**)&buf_a, numbytes);
  hipMalloc((void**)&buf_res, numbytes);
  hipMalloc((void**)&buf_value, sizeof(int));
  //从内存拷贝至显存
  hipMemcpy(buf_a, a, numbytes, hipMemcpyHostToDevice);
  hipMemcpy(buf_value, &value, sizeof(int), hipMemcpyHostToDevice);

  _restricted_multiply<<<1,numbytes>>>(buf_value, buf_a,  buf_res, numbytes);
  hipMemcpy(res, buf_res, numbytes, hipMemcpyDeviceToHost);
  return;
}

void test_xor(uint8_t * a, uint8_t * b, uint8_t * res, int numbytes){
  uint8_t *buf_a;
  uint8_t *buf_b;
  uint8_t *buf_res;

  hipMalloc((void**)&buf_a, numbytes);
  hipMalloc((void**)&buf_b, numbytes);
  hipMalloc((void**)&buf_res, numbytes);
  //从内存拷贝至显存
  hipMemcpy(buf_a, a, numbytes, hipMemcpyHostToDevice);
  hipMemcpy(buf_b, b, numbytes, hipMemcpyHostToDevice);

  _xor<<<1, numbytes>>>(buf_a, buf_b, buf_res, numbytes);
  hipMemcpy(res, buf_res, numbytes, hipMemcpyDeviceToHost);
  return;
}

//uint8_t * seed_0, uint8_t * seed_1分别是初始化后的随机数种子
//uint8_t * generated_value_cpu_0是表示给party0生成的随机数结果存放位置， uint8_t * generated_value_cpu_1是表示给party1生成的随机数结果存放位置
void fss_generate(uint8_t * r, uint8_t * seed0, uint8_t * seed1, uint8_t * key, uint8_t * generated_value_cpu_0, uint8_t * generated_value_cpu_1, int numbytes){
    if(numbytes!=8 && numbytes!=16){
      printf("only support 64 or 128 bits");
      return;
    }
    uint8_t * generated_value_0;
    uint8_t * generated_value_1;
    uint8_t * w_d;
    uint8_t * w;
    
    hipMemcpyToSymbol(HIP_SYMBOL(sbox), sbox, sizeof(uint8_t)*256);
   

    int bit_length = numbytes * 8;
    //分配扩展密钥
    w = (uint8_t*)malloc(240*sizeof(uint8_t));
    aes_key_expansion(key, w);

    hipMalloc((void**)&w_d, 240*sizeof(uint8_t));

    // //分配输出的generated_value的长度
    hipMalloc((void**)&generated_value_0, (bit_length - 1) * 2 * (numbytes + 1) + numbytes);
    hipMalloc((void**)&generated_value_1, (bit_length - 1) * 2 * (numbytes + 1) + numbytes);
    //分配数据结构空间
    
     if(numbytes = 8){
      fss_gen_struct_64 * fss_gen = {nullptr};
      hipMalloc((void**)&fss_gen, sizeof(fss_gen_struct_64));
      // hipMalloc((void**)&fss_gen->va, numbytes);
      // hipMalloc((void**)&(fss_gen->keep), sizeof(int));
      // hipMalloc((void**)&(fss_gen->lose), sizeof(int));
      // for(int idx = 0 ; idx < 2 ; idx++){
      //     hipMalloc((void**)&fss_gen->seed[idx], numbytes);
      //     hipMalloc((void**)&fss_gen->scw[idx], numbytes);
      //     hipMalloc((void**)&fss_gen->vcw[idx], numbytes);
      //     hipMalloc((void**)&fss_gen->tcw[idx], 1);
      //     hipMalloc((void**)&fss_gen->convert[idx], numbytes);
      //     hipMalloc((void**)&fss_gen->convert_seed[idx], numbytes);
      //     hipMalloc((void**)&fss_gen->inter_val[idx], 2*numbytes+1);
      //     for(int jdx = 0; jdx < 2; jdx++){
      //         hipMalloc((void**)&fss_gen->s[idx][jdx], numbytes);
      //         hipMalloc((void**)&fss_gen->v[idx][jdx], numbytes);
      //         hipMalloc((void**)&fss_gen->t[idx][jdx], 1);
      //         hipMalloc((void**)&fss_gen->pre_t[idx][jdx], 1);
      //     }
      // }
      // // //初始化种子
      // hipMemcpy(w_d, w, 240*sizeof(uint8_t), hipMemcpyHostToDevice);
      // hipMemcpy(fss_gen->r, r, numbytes, hipMemcpyHostToDevice);
      // hipMemcpy(fss_gen->seed[0], seed0, numbytes, hipMemcpyHostToDevice);
      // hipMemcpy(fss_gen->seed[1], seed1, numbytes, hipMemcpyHostToDevice);
      // hipMemset(fss_gen->va, 0, hipMemcpyHostToDevice);
      // hipMemset(fss_gen->pre_t[0], 0, hipMemcpyHostToDevice);
      // hipMemset(fss_gen->pre_t[1], 1, hipMemcpyHostToDevice);
        printf("space prepared!");
        dim3 dimBlock(ceil((double)numbytes / (double)(THREADS_PER_BLOCK * AES_BLOCK_SIZE)));
        dim3 dimGrid(THREADS_PER_BLOCK);

        fss_generate_gpu_64<<<dimBlock, dimGrid>>>(fss_gen, w_d, generated_value_cpu_0, generated_value_1, numbytes);
    }
    else{
      fss_gen_struct_128 * fss_gen = {nullptr};
      hipMalloc((void**)&fss_gen, sizeof(fss_gen_struct_128));
      dim3 dimBlock(ceil((double)numbytes / (double)(THREADS_PER_BLOCK * AES_BLOCK_SIZE)));
      dim3 dimGrid(THREADS_PER_BLOCK);
      fss_generate_gpu_128<<<dimBlock, dimGrid>>>(fss_gen, w_d, generated_value_cpu_0, generated_value_1, numbytes);
    }


}

