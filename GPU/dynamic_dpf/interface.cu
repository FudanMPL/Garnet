#include "hip/hip_runtime.h"
#include "aes.cu"
#include "utils.cu"
#include "fss_struct.h"
#include "fss.cu"
#include "test.cu"
#include <thrust/device_vector.h>
#include <cmath>

void fss_dpf_generate(InputByteRelatedValuesGen cpu_values, aes_gen_block * cpu_aes_block_array, int input_length, int parallel){
    int lambda = 127;  
    int input_byte = ceil(input_length/8);  
    BYTE key[240];
    int keyLen = 16;
    int blockLen = 16;
    KeyBlock * cuda_key_block;
    hipMalloc(&cuda_key_block, 2 * sizeof(class KeyBlock));
    prepare_key(cuda_key_block, key, keyLen);

    // printGpuBytes<<<1,1>>>(cuda_key_block->cuda_key[0], 0, 240);
    int thrdperblock, num_sm;
    init_sm_thrd(num_sm, thrdperblock, parallel);
    std::cout << num_sm << " " << thrdperblock <<  std::endl;

    dim3 ThreadperBlock(thrdperblock);
    dim3 BlockperGrid(num_sm);


    FssDpfGen * cuda_dpf_gen;   
    aes_gen_block * cuda_aes_block_array;
    
    // input length related values
    // random values, shape = [parallel, input_byte]
    uint8_t * cuda_r;
    hipMalloc(&cuda_r, parallel * input_byte * sizeof(uint8_t));
    hipMemcpy(cuda_r, cpu_values.r, parallel * input_byte * sizeof(uint8_t), hipMemcpyHostToDevice);
    // correction words, scw.shape = [parallel, input_length, input_byte]
    uint8_t * cuda_scw;
    hipMalloc(&cuda_scw, parallel * input_length * LAMBDA_BYTE * sizeof(uint8_t));
    // tcw.shape = [parallel, input_length]
    bool * cuda_tcw_0;
    bool * cuda_tcw_1;
    hipMalloc(&cuda_tcw_0, parallel * input_length * sizeof(bool));
    hipMalloc(&cuda_tcw_1, parallel * input_length * sizeof(bool));
    // output.shape = [parallel, input_byte]
    uint8_t * cuda_output;
    hipMalloc(&cuda_output, parallel * input_byte * sizeof(uint8_t));
    hipMalloc(&cuda_dpf_gen, parallel*sizeof(class FssDpfGen));
    hipMalloc(&cuda_aes_block_array, parallel*sizeof(class aes_gen_block));
    hipMemcpy(cuda_aes_block_array, cpu_aes_block_array, parallel*sizeof(class aes_gen_block), hipMemcpyHostToDevice);
    
    hipDeviceSynchronize();
    hipEvent_t start1;
    hipEventCreate(&start1);
    hipEvent_t stop1;
    hipEventCreate(&stop1);
    hipEventRecord(start1);

    gen_init<<<BlockperGrid, ThreadperBlock>>>(cuda_dpf_gen, parallel);
    for(int i = 0; i < input_length; i++){        
        for(int j = 0; j < 2; j++){
            AES_Encrypt_Gen<<<BlockperGrid,ThreadperBlock>>>(cuda_aes_block_array, cuda_key_block, 176, j, parallel);
            st_copy_gen<<<BlockperGrid,ThreadperBlock>>>(cuda_aes_block_array, cuda_dpf_gen, j, parallel); 
        }       
        cw_update_gen<<<BlockperGrid, ThreadperBlock>>>(cuda_r, cuda_scw, cuda_tcw_0, cuda_tcw_1, cuda_dpf_gen, i, input_byte, input_length, parallel);        
        for(int b = 0; b < 2; b++){
            st_update_gen<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_scw, cuda_tcw_0, cuda_tcw_1, cuda_dpf_gen, i, b, input_byte, input_length, parallel);
        }
    }
    final_cw_update_gen<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_output, cuda_dpf_gen, input_byte, parallel);
    hipMemcpy(cpu_values.scw, cuda_scw, parallel * input_length * LAMBDA_BYTE * sizeof(uint8_t), hipMemcpyDeviceToHost);
    hipMemcpy(cpu_values.tcw[0], cuda_tcw_0, parallel * input_length * sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(cpu_values.tcw[1], cuda_tcw_1, parallel * input_length * sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(cpu_values.output, cuda_output, parallel * input_byte * sizeof(uint8_t), hipMemcpyDeviceToHost);

    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    float msecTotal1,total;
    hipEventElapsedTime(&msecTotal1, start1, stop1);
    total=msecTotal1/1000;
    printf("eval time:%f\n",total);
    hipFree(cuda_dpf_gen);
    hipFree(cuda_aes_block_array);
    hipFree(cuda_r);
    hipFree(cuda_output);
    hipFree(cuda_scw);
    hipFree(cuda_tcw_0);
    hipFree(cuda_tcw_1);
}

void fss_dpf_evaluate(InputByteRelatedValuesEval cpu_eval_values, InputByteRelatedValuesGen cpu_values, aes_eval_block * cpu_aes_block_array, bool party, int input_length, int parallel){
    int lambda = 127;
    int input_byte = ceil(input_length/8);  
    
    BYTE key[240];
    int keyLen = 16;
    int blockLen = 16;
    KeyBlock * cuda_key_block;
    hipMalloc(&cuda_key_block, 2 * sizeof(class KeyBlock));
    prepare_key(cuda_key_block, key, keyLen); 

    int thrdperblock, num_sm;
    init_sm_thrd(num_sm, thrdperblock, parallel);
    std::cout << num_sm << " " << thrdperblock <<  std::endl;

    dim3 ThreadperBlock(thrdperblock);
    dim3 BlockperGrid(num_sm);

    
    FssDpfEval * cuda_dpf_eval;   
    aes_eval_block * cuda_aes_block_array;
    uint8_t * cuda_reveal;
    uint8_t * cuda_scw;
    bool * cuda_tcw_0;
    bool * cuda_tcw_1;
    uint8_t * cuda_output;
    uint8_t * cuda_result;

    // input length related values
    // random values, shape = [parallel, input_byte]
    
    hipMalloc(&cuda_reveal, parallel * input_byte * sizeof(uint8_t));
    hipMemcpy(cuda_reveal, cpu_values.r, parallel * input_byte * sizeof(uint8_t), hipMemcpyHostToDevice);
    // correction words, scw.shape = [parallel, input_length, LAMBDA_BYTE]
    hipMalloc(&cuda_scw, parallel * input_length * LAMBDA_BYTE * sizeof(uint8_t));
    hipMemcpy(cuda_scw, cpu_values.scw, parallel * input_length * LAMBDA_BYTE * sizeof(uint8_t), hipMemcpyHostToDevice);
    // tcw.shape = [parallel, input_length]
    hipMalloc(&cuda_tcw_0, parallel * input_length * sizeof(bool));
    hipMemcpy(cuda_tcw_0, cpu_values.tcw[0], parallel * input_length * sizeof(bool), hipMemcpyHostToDevice);
    hipMalloc(&cuda_tcw_1, parallel * input_length * sizeof(bool));
    hipMemcpy(cuda_tcw_1, cpu_values.tcw[1], parallel * input_length * sizeof(bool), hipMemcpyHostToDevice);
    // output.shape = [parallel, input_byte]
    hipMalloc(&cuda_output, parallel * input_byte * sizeof(uint8_t));
    hipMemcpy(cuda_output, cpu_values.output, parallel * input_byte * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMalloc(&cuda_result, parallel * input_byte * sizeof(uint8_t));
    hipMalloc(&cuda_dpf_eval, parallel*sizeof(class FssDpfEval));
    hipMalloc(&cuda_aes_block_array, parallel*sizeof(class aes_eval_block));
    hipMemcpy(cuda_aes_block_array, cpu_aes_block_array, parallel*sizeof(class aes_eval_block), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    hipEvent_t start1;
    hipEventCreate(&start1);
    hipEvent_t stop1;
    hipEventCreate(&stop1);
    hipEventRecord(start1);
    eval_init<<<BlockperGrid, ThreadperBlock>>>(cuda_dpf_eval, party, parallel);
    for(int i = 0; i < input_length; i++){
        AES_Encrypt_Eval<<<BlockperGrid,ThreadperBlock>>>(cuda_aes_block_array, cuda_key_block, 176, parallel);
        st_init_eval<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_scw, cuda_dpf_eval, i, input_byte, input_length, parallel);
        st_update_eval<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_reveal, cuda_scw, cuda_tcw_0, cuda_tcw_1, cuda_dpf_eval, i, input_byte, input_length, parallel);  
    }
    result_update_eval<<<BlockperGrid, ThreadperBlock>>>(cuda_result, cuda_aes_block_array, cuda_output, cuda_dpf_eval, input_byte, parallel);
    hipMemcpy(cpu_eval_values.result, cuda_result, parallel*input_byte*sizeof(uint8_t), hipMemcpyDeviceToHost);
    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    float msecTotal1,total;
    hipEventElapsedTime(&msecTotal1, start1, stop1);
    total=msecTotal1/1000;
    printf("eval time:%f\n",total);
    hipFree(cuda_dpf_eval);
    hipFree(cuda_key_block);
    hipFree(cuda_reveal);
    hipFree(cuda_result);
    hipFree(cuda_scw);
    hipFree(cuda_tcw_0);
    hipFree(cuda_tcw_1);
    hipFree(cuda_output);
    hipFree(cuda_aes_block_array);
}

void fss_dpf_generate(InputByteRelatedValuesGen cpu_values, aes_gen_block * cpu_aes_block_array, int input_length, int parallel){
    int lambda = 127;  
    int input_byte = ceil(input_length/8);  
    BYTE key[240];
    int keyLen = 16;
    int blockLen = 16;
    KeyBlock * cuda_key_block;
    hipMalloc(&cuda_key_block, 2 * sizeof(class KeyBlock));
    prepare_key(cuda_key_block, key, keyLen);

    // printGpuBytes<<<1,1>>>(cuda_key_block->cuda_key[0], 0, 240);
    int thrdperblock, num_sm;
    init_sm_thrd(num_sm, thrdperblock, parallel);
    std::cout << num_sm << " " << thrdperblock <<  std::endl;

    dim3 ThreadperBlock(thrdperblock);
    dim3 BlockperGrid(num_sm);


    FssDpfGen * cuda_dpf_gen;   
    aes_gen_block * cuda_aes_block_array;
    
    // input length related values
    // random values, shape = [parallel, input_byte]
    uint8_t * cuda_r;
    hipMalloc(&cuda_r, parallel * input_byte * sizeof(uint8_t));
    hipMemcpy(cuda_r, cpu_values.r, parallel * input_byte * sizeof(uint8_t), hipMemcpyHostToDevice);
    // correction words, scw.shape = [parallel, input_length, input_byte]
    uint8_t * cuda_scw;
    hipMalloc(&cuda_scw, parallel * input_length * LAMBDA_BYTE * sizeof(uint8_t));
    // tcw.shape = [parallel, input_length]
    bool * cuda_tcw_0;
    bool * cuda_tcw_1;
    hipMalloc(&cuda_tcw_0, parallel * input_length * sizeof(bool));
    hipMalloc(&cuda_tcw_1, parallel * input_length * sizeof(bool));
    // output.shape = [parallel, input_byte]
    uint8_t * cuda_output;
    hipMalloc(&cuda_output, parallel * input_byte * sizeof(uint8_t));
    hipMalloc(&cuda_dpf_gen, parallel*sizeof(class FssDpfGen));
    hipMalloc(&cuda_aes_block_array, parallel*sizeof(class aes_gen_block));
    hipMemcpy(cuda_aes_block_array, cpu_aes_block_array, parallel*sizeof(class aes_gen_block), hipMemcpyHostToDevice);
    
    hipDeviceSynchronize();
    hipEvent_t start1;
    hipEventCreate(&start1);
    hipEvent_t stop1;
    hipEventCreate(&stop1);
    hipEventRecord(start1);

    gen_init<<<BlockperGrid, ThreadperBlock>>>(cuda_dpf_gen, parallel);
    for(int i = 0; i < input_length; i++){        
        for(int j = 0; j < 2; j++){
            AES_Encrypt_Gen<<<BlockperGrid,ThreadperBlock>>>(cuda_aes_block_array, cuda_key_block, 176, j, parallel);
            st_copy_gen<<<BlockperGrid,ThreadperBlock>>>(cuda_aes_block_array, cuda_dpf_gen, j, parallel); 
        }       
        cw_update_gen<<<BlockperGrid, ThreadperBlock>>>(cuda_r, cuda_scw, cuda_tcw_0, cuda_tcw_1, cuda_dpf_gen, i, input_byte, input_length, parallel);        
        for(int b = 0; b < 2; b++){
            st_update_gen<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_scw, cuda_tcw_0, cuda_tcw_1, cuda_dpf_gen, i, b, input_byte, input_length, parallel);
        }
    }
    final_cw_update_gen<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_output, cuda_dpf_gen, input_byte, parallel);
    hipMemcpy(cpu_values.scw, cuda_scw, parallel * input_length * LAMBDA_BYTE * sizeof(uint8_t), hipMemcpyDeviceToHost);
    hipMemcpy(cpu_values.tcw[0], cuda_tcw_0, parallel * input_length * sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(cpu_values.tcw[1], cuda_tcw_1, parallel * input_length * sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(cpu_values.output, cuda_output, parallel * input_byte * sizeof(uint8_t), hipMemcpyDeviceToHost);

    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    float msecTotal1,total;
    hipEventElapsedTime(&msecTotal1, start1, stop1);
    total=msecTotal1/1000;
    printf("eval time:%f\n",total);
    hipFree(cuda_dpf_gen);
    hipFree(cuda_aes_block_array);
    hipFree(cuda_r);
    hipFree(cuda_output);
    hipFree(cuda_scw);
    hipFree(cuda_tcw_0);
    hipFree(cuda_tcw_1);
}

void fss_dpf_evaluate(InputByteRelatedValuesEval cpu_eval_values, InputByteRelatedValuesGen cpu_values, aes_eval_block * cpu_aes_block_array, bool party, int input_length, int parallel){