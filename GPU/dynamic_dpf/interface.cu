#include "hip/hip_runtime.h"
#include "aes.cu"
#include "utils.cu"
#include "fss_struct.h"
#include "fss.cu"
#include "test.cu"
#include <cmath>

void fss_dpf_generate(RandomValueBlock * cpu_r_block, aes_gen_block * cpu_aes_block_array, CorrectionWord * cpu_cw, int parallel){
    int lambda = 127;
    int bit_length = INPUT_BYTE * 8;

    
    BYTE key[240];
    int keyLen = 16;
    int blockLen = 16;
    KeyBlock * cuda_key_block;
    hipMalloc(&cuda_key_block, 2 * sizeof(class KeyBlock));
    prepare_key(cuda_key_block, key, keyLen);

    // printGpuBytes<<<1,1>>>(cuda_key_block->cuda_key[0], 0, 240);
    int thrdperblock, num_sm;
    init_sm_thrd(num_sm, thrdperblock, parallel);
    std::cout << num_sm << " " << thrdperblock <<  std::endl;

    dim3 ThreadperBlock(thrdperblock);
    dim3 BlockperGrid(num_sm);


    CorrectionWord * cuda_cw;   
    FssDpfGen * cuda_dpf_gen;   
    RandomValueBlock * cuda_r_block;   
    aes_gen_block * cuda_aes_block_array;
    hipMalloc(&cuda_cw, parallel*sizeof(class CorrectionWord));
    hipMalloc(&cuda_dpf_gen, parallel*sizeof(class FssDpfGen));
    hipMalloc(&cuda_r_block, parallel*sizeof(class RandomValueBlock));
    hipMemcpy(cuda_r_block, cpu_r_block, parallel*sizeof(class RandomValueBlock), hipMemcpyHostToDevice);
    hipMalloc(&cuda_aes_block_array, parallel*sizeof(class aes_gen_block));
    hipMemcpy(cuda_aes_block_array, cpu_aes_block_array, parallel*sizeof(class aes_gen_block), hipMemcpyHostToDevice);
    
    hipDeviceSynchronize();
    hipEvent_t start1;
    hipEventCreate(&start1);
    hipEvent_t stop1;
    hipEventCreate(&stop1);
    hipEventRecord(start1);

    gen_init<<<BlockperGrid, ThreadperBlock>>>(cuda_dpf_gen, parallel);
    for(int i = 0; i < bit_length; i++){        
        for(int j = 0; j < 2; j++){
            AES_Encrypt_Gen<<<BlockperGrid,ThreadperBlock>>>(cuda_aes_block_array, cuda_key_block, 176, j, parallel);
            st_copy_gen<<<BlockperGrid,ThreadperBlock>>>(cuda_aes_block_array, cuda_dpf_gen, j, parallel); 
        }
        cw_update_gen<<<BlockperGrid, ThreadperBlock>>>(cuda_r_block, cuda_cw, cuda_dpf_gen, i, parallel);        
        
        for(int b = 0; b < 2; b++){
            st_update_gen<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_cw, cuda_dpf_gen, i, b, parallel);
        }
    }
    final_cw_update_gen<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_cw, cuda_dpf_gen, parallel);

    hipMemcpy(cpu_cw, cuda_cw, parallel*sizeof(class CorrectionWord), hipMemcpyDeviceToHost);

    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    float msecTotal1,total;
    hipEventElapsedTime(&msecTotal1, start1, stop1);
    total=msecTotal1/1000;
    printf("eval time:%f\n",total);
    hipFree(cuda_dpf_gen);
    hipFree(cuda_key_block);
    hipFree(cuda_cw);
    hipFree(cuda_r_block);
    hipFree(cuda_aes_block_array);
}


void fss_dpf_evaluate(RevealValueBlock * cpu_reveal, aes_eval_block * cpu_aes_block_array, CorrectionWord * cpu_cw, ResultBlock * cpu_res, bool party, int parallel){
    int lambda = 127;
    int bit_length = INPUT_BYTE * 8;

    
    BYTE key[240];
    int keyLen = 16;
    int blockLen = 16;
    KeyBlock * cuda_key_block;
    hipMalloc(&cuda_key_block, 2 * sizeof(class KeyBlock));
    prepare_key(cuda_key_block, key, keyLen); 

    int thrdperblock, num_sm;
    init_sm_thrd(num_sm, thrdperblock, parallel);
    std::cout << num_sm << " " << thrdperblock <<  std::endl;

    dim3 ThreadperBlock(thrdperblock);
    dim3 BlockperGrid(num_sm);

    CorrectionWord * cuda_cw;   
    FssDpfEval * cuda_dpf_eval;   
    RevealValueBlock * cuda_reveal_block;   
    aes_eval_block * cuda_aes_block_array;
    ResultBlock * cuda_res;
    hipMalloc(&cuda_cw, parallel*sizeof(class CorrectionWord));
    hipMemcpy(cuda_cw, cpu_cw, parallel*sizeof(class CorrectionWord), hipMemcpyHostToDevice);
    hipMalloc(&cuda_dpf_eval, parallel*sizeof(class FssDpfEval));
    hipMalloc(&cuda_reveal_block, parallel*sizeof(class RevealValueBlock));
    hipMemcpy(cuda_reveal_block, cpu_reveal, parallel*sizeof(class RevealValueBlock), hipMemcpyHostToDevice);
    hipMalloc(&cuda_aes_block_array, parallel*sizeof(class aes_eval_block));
    hipMemcpy(cuda_aes_block_array, cpu_aes_block_array, parallel*sizeof(class aes_eval_block), hipMemcpyHostToDevice);
    hipMalloc(&cuda_res, parallel*sizeof(class ResultBlock));

    hipDeviceSynchronize();
    hipEvent_t start1;
    hipEventCreate(&start1);
    hipEvent_t stop1;
    hipEventCreate(&stop1);
    hipEventRecord(start1);
    eval_init<<<BlockperGrid, ThreadperBlock>>>(cuda_dpf_eval, party, parallel);
    for(int i = 0; i < bit_length; i++){
        AES_Encrypt_Eval<<<BlockperGrid,ThreadperBlock>>>(cuda_aes_block_array, cuda_key_block, 176, parallel);
        test<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_dpf_eval, parallel);
        st_init_eval<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_cw, cuda_dpf_eval, i, parallel);
        st_update_eval<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_reveal_block, cuda_cw, cuda_dpf_eval, i, parallel);  
    }
    result_update_eval<<<BlockperGrid, ThreadperBlock>>>(cuda_res, cuda_aes_block_array, cuda_cw, cuda_dpf_eval, parallel);
    hipMemcpy(cpu_res, cuda_res, parallel*sizeof(class ResultBlock), hipMemcpyDeviceToHost);
    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    float msecTotal1,total;
    hipEventElapsedTime(&msecTotal1, start1, stop1);
    total=msecTotal1/1000;
    printf("eval time:%f\n",total);
    hipFree(cuda_dpf_eval);
    hipFree(cuda_key_block);
    hipFree(cuda_cw);
    hipFree(cuda_reveal_block);
    hipFree(cuda_aes_block_array);
}

void fss_dpf_compress_generate(InputByteRelatedValuesGen cpu_values, aes_gen_block * cpu_aes_block_array, int input_length, int parallel){
    int lambda = 127;  
    int input_byte = ceil(input_length/8);  
    BYTE key[240];
    int keyLen = 16;
    int blockLen = 16;
    KeyBlock * cuda_key_block;
    hipMalloc(&cuda_key_block, 2 * sizeof(class KeyBlock));
    prepare_key(cuda_key_block, key, keyLen);

    // printGpuBytes<<<1,1>>>(cuda_key_block->cuda_key[0], 0, 240);
    int thrdperblock, num_sm;
    init_sm_thrd(num_sm, thrdperblock, parallel);
    std::cout << num_sm << " " << thrdperblock <<  std::endl;

    dim3 ThreadperBlock(thrdperblock);
    dim3 BlockperGrid(num_sm);


    FssDpfGen * cuda_dpf_gen;   
    RandomValueBlock * cuda_r_block;   
    aes_gen_block * cuda_aes_block_array;
    
    // input length related values
    // random values, shape = [parallel, input_byte]
    uint8_t * cuda_r;
    hipMalloc(&cuda_r, parallel * input_byte * sizeof(uint8_t));
    hipMemcpy(cuda_r, cpu_values.r, parallel * input_byte * sizeof(uint8_t), hipMemcpyHostToDevice);
    // correction words, scw.shape = [parallel, input_length, input_byte]
    uint8_t * cuda_scw;
    hipMalloc(&cuda_scw, parallel * input_length * LAMBDA_BYTE * sizeof(uint8_t));
    // tcw.shape = [parallel, input_length]
    bool * cuda_tcw_0;
    bool * cuda_tcw_1;
    hipMalloc(&cuda_tcw_0, parallel * input_length * sizeof(bool));
    hipMalloc(&cuda_tcw_1, parallel * input_length * sizeof(bool));
    // output.shape = [parallel, input_byte]
    uint8_t * cuda_output;
    hipMalloc(&cuda_output, parallel * input_byte * sizeof(uint8_t));
    hipMalloc(&cuda_dpf_gen, parallel*sizeof(class FssDpfGen));
    hipMalloc(&cuda_aes_block_array, parallel*sizeof(class aes_gen_block));
    hipMemcpy(cuda_aes_block_array, cpu_aes_block_array, parallel*sizeof(class aes_gen_block), hipMemcpyHostToDevice);
    
    hipDeviceSynchronize();
    hipEvent_t start1;
    hipEventCreate(&start1);
    hipEvent_t stop1;
    hipEventCreate(&stop1);
    hipEventRecord(start1);

    gen_init<<<BlockperGrid, ThreadperBlock>>>(cuda_dpf_gen, parallel);
    for(int i = 0; i < input_length; i++){        
        for(int j = 0; j < 2; j++){
            AES_Encrypt_Gen<<<BlockperGrid,ThreadperBlock>>>(cuda_aes_block_array, cuda_key_block, 176, j, parallel);
            st_copy_gen<<<BlockperGrid,ThreadperBlock>>>(cuda_aes_block_array, cuda_dpf_gen, j, parallel); 
        }
        test_cw_update_gen<<<BlockperGrid, ThreadperBlock>>>(cuda_r, cuda_scw, cuda_tcw_0, cuda_tcw_1, cuda_dpf_gen, i, input_byte, parallel);        
        
        for(int b = 0; b < 2; b++){
            test_st_update_gen<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_scw, cuda_tcw_0, cuda_tcw_1, cuda_dpf_gen, i, b, input_byte, parallel);
        }
    }
    test_final_cw_update_gen<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_output, cuda_dpf_gen, input_byte, parallel);
    hipMemcpy(cpu_values.scw, cuda_scw, parallel * input_length * LAMBDA_BYTE * sizeof(uint8_t), hipMemcpyDeviceToHost);
    hipMemcpy(cpu_values.tcw[0], cuda_tcw_0, parallel * input_length * sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(cpu_values.tcw[1], cuda_tcw_1, parallel * input_length * sizeof(bool), hipMemcpyDeviceToHost);

    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    float msecTotal1,total;
    hipEventElapsedTime(&msecTotal1, start1, stop1);
    total=msecTotal1/1000;
    printf("eval time:%f\n",total);
    hipFree(cuda_dpf_gen);
    hipFree(cuda_aes_block_array);
    hipFree(cuda_r);
    hipFree(cuda_output);
    hipFree(cuda_scw);
    hipFree(cuda_tcw_0);
    hipFree(cuda_tcw_1);
}

// void fss_dpf_compress_evaluate(InputByteRelatedValuesEval cpu_values, aes_eval_block * cpu_aes_block_array, ResultBlock * cpu_res, bool party, int input_length, int parallel){
//     int lambda = 127;
//     int input_byte = ceil(input_length/8);  
    
//     BYTE key[240];
//     int keyLen = 16;
//     int blockLen = 16;
//     KeyBlock * cuda_key_block;
//     hipMalloc(&cuda_key_block, 2 * sizeof(class KeyBlock));
//     prepare_key(cuda_key_block, key, keyLen); 

//     int thrdperblock, num_sm;
//     init_sm_thrd(num_sm, thrdperblock, parallel);
//     std::cout << num_sm << " " << thrdperblock <<  std::endl;

//     dim3 ThreadperBlock(thrdperblock);
//     dim3 BlockperGrid(num_sm);

    
//     FssDpfEval * cuda_dpf_eval;   
//     aes_eval_block * cuda_aes_block_array;
//     uint8_t * cuda_reveal;
//     uint8_t * cuda_scw;
//     bool * cuda_tcw_0;
//     bool * cuda_tcw_1;
//     uint8_t * cuda_output;
//     uint8_t * cuda_result;

//     hipMalloc(&cuda_cw, parallel*sizeof(class CorrectionWord));
//     hipMemcpy(cuda_cw, cpu_cw, parallel*sizeof(class CorrectionWord), hipMemcpyHostToDevice);
//     hipMalloc(&cuda_dpf_eval, parallel*sizeof(class FssDpfEval));
//     hipMalloc(&cuda_reveal_block, parallel*sizeof(class RevealValueBlock));
//     hipMemcpy(cuda_reveal_block, cpu_reveal, parallel*sizeof(class RevealValueBlock), hipMemcpyHostToDevice);
//     hipMalloc(&cuda_aes_block_array, parallel*sizeof(class aes_eval_block));
//     hipMemcpy(cuda_aes_block_array, cpu_aes_block_array, parallel*sizeof(class aes_eval_block), hipMemcpyHostToDevice);
//     hipMalloc(&cuda_res, parallel*sizeof(class ResultBlock));

//     hipDeviceSynchronize();
//     hipEvent_t start1;
//     hipEventCreate(&start1);
//     hipEvent_t stop1;
//     hipEventCreate(&stop1);
//     hipEventRecord(start1);
//     eval_init<<<BlockperGrid, ThreadperBlock>>>(cuda_dpf_eval, party, parallel);
//     for(int i = 0; i < bit_length; i++){
//         AES_Encrypt_Eval<<<BlockperGrid,ThreadperBlock>>>(cuda_aes_block_array, cuda_key_block, 176, parallel);
//         test<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_dpf_eval, parallel);
//         st_init_eval<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_cw, cuda_dpf_eval, i, parallel);
//         st_update_eval<<<BlockperGrid, ThreadperBlock>>>(cuda_aes_block_array, cuda_reveal_block, cuda_cw, cuda_dpf_eval, i, parallel);  
//     }
//     result_update_eval<<<BlockperGrid, ThreadperBlock>>>(cuda_res, cuda_aes_block_array, cuda_cw, cuda_dpf_eval, parallel);
//     hipMemcpy(cpu_res, cuda_res, parallel*sizeof(class ResultBlock), hipMemcpyDeviceToHost);
//     hipEventRecord(stop1);
//     hipEventSynchronize(stop1);
//     float msecTotal1,total;
//     hipEventElapsedTime(&msecTotal1, start1, stop1);
//     total=msecTotal1/1000;
//     printf("eval time:%f\n",total);
//     hipFree(cuda_dpf_eval);
//     hipFree(cuda_key_block);
//     hipFree(cuda_cw);
//     hipFree(cuda_reveal_block);
//     hipFree(cuda_aes_block_array);
// }